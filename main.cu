#include "hip/hip_runtime.h"
// #pragma once
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include <iomanip>      // std::setprecision

#include ""

#include "input.h"

using namespace std;

#define NUM_THREADS_PER_BLOCK 512

int* create_shifts (char* pattern);

int linear_horspool_match (char* text, char* pattern, int* shift_table, unsigned int* num_matches, int chunk_size,
    int num_chunks, int text_size, int pat_len, int myId);

__global__ void horspool_match (char* text, char* pattern, int* shift_table, unsigned int* num_matches, int chunk_size,
    int num_chunks, int text_size, int pat_len);
__global__ void prescan(int *g_odata, int *g_idata, int n);

int determineNumBlocks(vector<string_chunk> chunks) {
	int numBlocks = 0;
	for (int i = 0; i < chunks.size(); i = i + NUM_THREADS_PER_BLOCK) {
		numBlocks++;
	}
	return numBlocks;
}

/*
 *  Driver function
 *  argv[0] is target pattern string
 *  argv[1] is text path
 */
int main(int argc, char* argv[])
{
    const int TABLE_SIZ = 126;
    int target_len = 0;

    if (argc == 2 && (strcmp(argv[1], "-h") || strcmp(argv[1], "--help"))){
        cout << "`match.exe` finds exact matches to a target string in text files." << endl
            << "Type ./main.exe {target_string} {text file path} to use the program." << endl
            << "Text file paths must be relative to the directory of `main.exe`." << endl;
        exit(0);
    } else if (argc == 1) {
        cout << "ERROR: Please pass in a target string and a file path." << endl;
        exit(-1);
    }
    for (int i = 1; i < argc - 1; ++i) {
        target_len += strlen(argv[i]);
    }
    target_len += argc - 3;
    

	Input inputObj(argv[argc - 1]);
    char* flatText = inputObj.flattenText();

    char* testPattern = (char*)malloc(target_len * sizeof(char) + 1);
    string input = argv[1];
    if (argc > 3) {
        for (int i = 2; i < argc - 1; ++i) {
            input = input + " " + argv[i];

        }
    }

    strcpy (testPattern, input.data());
    testPattern[target_len] = '\0';

    int* skipTable = create_shifts(testPattern);
	unsigned int* numMatches = (unsigned int*)malloc(1 * sizeof(unsigned int));
	*numMatches = 0;

	int fullTextSize = inputObj.getChunks().size() * CHUNK_SIZE * sizeof(char);
	int patternSize = strlen(testPattern) * sizeof(char);
	int skipTableSize = TABLE_SIZ * sizeof(int);

	char* d_fullText;
	char* d_testPattern;
	int* d_skipTable;
	unsigned int* d_numMatches;
    unsigned int* parallel_result = (unsigned int*) malloc(sizeof(unsigned int));

	hipMalloc((void**)& d_fullText, fullTextSize);
	hipMalloc((void**)& d_testPattern, patternSize);
	hipMalloc((void**)& d_skipTable, skipTableSize);
	hipMalloc((void**)& d_numMatches, sizeof(unsigned int));

	hipMemcpy(d_fullText, flatText, fullTextSize, hipMemcpyHostToDevice);
	hipMemcpy(d_testPattern, testPattern, patternSize, hipMemcpyHostToDevice);
	hipMemcpy(d_skipTable, skipTable, skipTableSize, hipMemcpyHostToDevice);
	hipMemcpy(d_numMatches, numMatches, sizeof(unsigned int), hipMemcpyHostToDevice);

    
    time_t start, end, start1, end1 = 0;
    int text_len = strlen(flatText);
    int pat_len = strlen(testPattern); 
    int num_chunks = inputObj.getChunks().size();
    int numBlocks = determineNumBlocks(inputObj.getChunks());
    hipDeviceSynchronize();

    time(&start);   
    start = clock();

	horspool_match << <numBlocks, NUM_THREADS_PER_BLOCK, NUM_THREADS_PER_BLOCK * sizeof(int) >> > (d_fullText, d_testPattern, d_skipTable, d_numMatches, CHUNK_SIZE, 
        num_chunks, text_len, pat_len);
        hipDeviceSynchronize();
    
    hipMemcpy(parallel_result, d_numMatches, sizeof(unsigned int), hipMemcpyDeviceToHost);
    end = clock();
    

    start1 = clock();
    unsigned int result = 0;
    for(int myId =0; myId < numBlocks * NUM_THREADS_PER_BLOCK; myId++){
        result += linear_horspool_match(flatText, testPattern, skipTable, numMatches, CHUNK_SIZE, 
            num_chunks, text_len, pat_len, myId);    
    }
    end1 = clock();
    hipDeviceSynchronize();

    // Calculating total time taken by the program. 
    double time_taken = double(end - start)/ CLOCKS_PER_SEC; 
    cout << "Time taken by parallel program: " << setprecision(9) << time_taken << " secs." << endl;
    cout << "There are " << *parallel_result << " exact matches to string `" << input << "`" << 
        endl << "found by parallel program in file `" << argv[argc-1] <<"`"<< endl << endl;

    time_taken = double(end1 - start1)/ CLOCKS_PER_SEC;
    cout << "Time taken by linear program: " << setprecision(9) << time_taken <<  " secs." << endl; 
    cout << "There are " << result << " exact matches to string `" << input << "`" <<
        endl << "found by linear program in file `" << argv[argc-1] <<"`"<< endl;

    hipFree(d_fullText);
    hipFree(d_testPattern);
    hipFree(d_skipTable);
    hipFree(d_numMatches);

	free(testPattern);
	free(skipTable);
    free(numMatches);
}

int linear_horspool_match (char* text, char* pattern, int* shift_table, unsigned int* num_matches, int chunk_size,
    int num_chunks, int text_size, int pat_len, int myId) {
        
        const int TABLE_SIZ = 126;

        int count = 0;
        int text_length = (chunk_size * myId) + chunk_size + pat_len - 1;
    
        // don't need to check first pattern_length - 1 characters
        int i = (myId*chunk_size) + pat_len - 1;
        int k = 0;
        while(i < text_length) {
            // reset matched character count
            k = 0;
    
            if (i >= text_size) {
            // break out if i tries to step past text length
                break;
            }

            if (text[i] >= TABLE_SIZ || text[i] < 0) {
                // move to next char if unknown char (Unicode, etc.)
                ++i;
            } else {
                while(k <= pat_len - 1 && pattern[pat_len - 1 - k] == text[i - k]) {
                // increment matched character count
                    k++;
                }
                if(k == pat_len) {
                // increment pattern count, text index
                    ++count;
                    ++i;
        
                } else {
                    // add on shift if known char
                    i = i + shift_table[text[i]];
                }
            }
        }
        return count;
        // Add count to total matches atomically
    
    }
    

/**
 *  Purpose:
 *    Boyer-Moore-Horspool pattern matching algorithm implementation
 * 
 *  Args:
 *    text        {char*}: Text c-string - still text
 *    pattern     {char*}: Target c-string - still pattern
 *    shift_table  {int*}: Skip table - shift table
 *    num_matches   {int}: Total match count - num_matches
 *    chunk_size    {int}: Length of chunk size
 *    num_chunks    {int}: Total number of chunks
 *    text_size     {int}: Integer text length
 *    pat_len       {int}: Integer pattern length
 *  Returns:
 *    None
 */ 
__global__ void horspool_match (char* text, char* pattern, int* shift_table, unsigned int* num_matches, int chunk_size,
    int num_chunks, int text_size, int pat_len) {
    
    const int TABLE_SIZ = 126;

    int count = 0;
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    if(myId > num_chunks){ //if thread is an invalid thread
        return;
    }

    int text_length = (chunk_size * myId) + chunk_size + pat_len - 1;

    // don't need to check first pattern_length - 1 characters
    int i = (myId*chunk_size) + pat_len - 1;
    int k = 0;
    while(i < text_length) {
        // reset matched character count
        k = 0;

        if (i >= text_size) {
        // break out if i tries to step past text length
            break;
        }

        if (text[i] >= TABLE_SIZ || text[i] < 0) {
            // move to next char if unknown char (Unicode, etc.)
            ++i;
        } else {
            while(k <= pat_len - 1 && pattern[pat_len - 1 - k] == text[i - k]) {
            // increment matched character count
                k++;
            }
            if(k == pat_len) {
            // increment pattern count, text index
                ++count;
                ++i;
    
            } else {
                // add on shift if known char
                i = i + shift_table[text[i]];
            }
        }
    }

    atomicAdd(num_matches, count);
}


/**
 *  Purpose:
 *    Create shift table for Boyer-Moore-Horspool algorithm
 *  
 *  Args:
 *    pattern {char*}: desired pattern c-string
 */ 
int* create_shifts (char* pattern)
{

    // Printable ASCII chars are 32-126 inclusive, line break is 10
    const int TABLE_SIZ = 126;

    const int FIRST_ASCII = 32;

    int length = strlen(pattern);
    int* shift_table = (int*) malloc (sizeof(int) * TABLE_SIZ);

    for(int i = 0; i < TABLE_SIZ; i++) {
        // set all entries to longest shift (pattern length)
        shift_table[i] = length;
    }
    for(int j = 0; j < length - 1; j++) {
        // set pattern characters to shortest shifts
        shift_table[pattern[j]] = length - 1 - j;
    }

    // assign shift of 1 for unprintable characters
    for (int i = 0; i < FIRST_ASCII; ++i) {
        shift_table[i] = 1;
    }

    return shift_table;
}